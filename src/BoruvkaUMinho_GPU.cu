#include "hip/hip_runtime.h"

#include "BoruvkaUMinho_GPU.cuh"


texture<unsigned int, 1, hipReadModeElementType> tex_psrc;
texture<unsigned int, 1, hipReadModeElementType> tex_outdegree;
texture<unsigned int, 1, hipReadModeElementType> tex_edgessrcdst;
texture<unsigned int, 1, hipReadModeElementType> tex_edgessrcwt;

__global__
void find_min_per_vertex(CSR_Graph g, unsigned int *vertex_minedge){
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id >= g.nnodes) return;

	unsigned min_edge = 0;
	unsigned min_weight = UINT_MAX;
	unsigned min_dst = g.nnodes;

	unsigned edge = tex1Dfetch(tex_psrc, id);
	unsigned last_edge = edge + tex1Dfetch(tex_outdegree, id);
	for(; edge < last_edge; ++edge)
	{
		unsigned wt = tex1Dfetch(tex_edgessrcwt, edge);
		unsigned dst = tex1Dfetch(tex_edgessrcdst, edge);
		//if(dst != g.nnodes)
		{
			if(wt < min_weight || (wt == min_weight && dst < min_dst))
			{
				min_weight = wt;
				min_edge = edge;
				min_dst = dst;
			}
		}
		//else if (dst == min_dst && wt >= min_weight)
		//{
		//	g.edgessrcdst[edge] = g.nnodes;
		//}
	}

	vertex_minedge[id] = min_edge;
}

__global__
void initialize_color(unsigned int nnodes, unsigned int *color, unsigned int *vertex_minedge){
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id >= nnodes) return;
	
	unsigned edge = vertex_minedge[id];
	if(edge == 0) color[id] = id;
	else color[id] = tex1Dfetch(tex_edgessrcdst, edge);
}

__global__
void propagate_color(unsigned int nnodes, unsigned int *color, unsigned int *changed){
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id >= nnodes) return;

	unsigned int my_color = color[id];
	unsigned int other_color = color[my_color];

	if(my_color != other_color)
	{
		color[id] = other_color;
		*changed = true;
	}
}

__global__
void remove_duplicates(CSR_Graph g, unsigned int *vertex_minedge){
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id >= g.nnodes) return;

	unsigned int edge = vertex_minedge[id];
	if(edge == 0) return;
	unsigned int dst = tex1Dfetch(tex_edgessrcdst, edge);

	unsigned int other_edge = vertex_minedge[dst];
	if(other_edge == 0) return;
	unsigned int other_dst = tex1Dfetch(tex_edgessrcdst, other_edge);

	if(id == other_dst && id > dst) // found loop and maintain edge by smaller vertex id
	{
		vertex_minedge[dst] = 0;
	}
}

__global__
void mark_mst_edges(unsigned int nnodes, unsigned int *selected_edges, unsigned int *vertex_minedge, unsigned int *map_edges){
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id >= nnodes) return;

	unsigned int edge = vertex_minedge[id];
	selected_edges[map_edges[edge]] = 1;
}



__global__
void create_new_vertex_id(CSR_Graph g, unsigned int *color, unsigned int *new_vertex, unsigned int *next_nnodes){
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id >= g.nnodes) return;

	if(id == color[id] && tex1Dfetch(tex_outdegree, id) > 0) // representative thread
	{	
		new_vertex[id] = 1;
	}
	else new_vertex[id] = 0;
}

__global__
void count_new_edges(CSR_Graph g, CSR_Graph next, unsigned int *color, unsigned int *new_vertex){
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id >= g.nnodes) return;

	unsigned my_color = color[id];
	// count how many edges I will be adding to supervertex
	unsigned new_edges = 0;
	unsigned edge = tex1Dfetch(tex_psrc, id);
	unsigned last_edge = edge + tex1Dfetch(tex_outdegree, id);

	for(; edge < last_edge; ++edge)
	{
		unsigned dst = tex1Dfetch(tex_edgessrcdst, edge);
		if(/*dst != g.nnodes &&*/ my_color != color[dst]) ++new_edges;
	}

	unsigned supervertex_id = new_vertex[my_color];
	atomicAdd(&(next.outdegree[supervertex_id]), new_edges);
}

__global__
void setup_psrc(CSR_Graph next, unsigned int *next_nedges){
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id >= next.nnodes) return;

	++next.psrc[id];
}

__global__
void insert_new_edges(CSR_Graph g, unsigned int *next_edgessrcdst, unsigned int *next_edgessrcwt, unsigned int *color, unsigned int *new_vertex, unsigned int *topedge_per_vertex, unsigned int *old_map_edges, unsigned int *new_map_edges){
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id >= g.nnodes) return;

	unsigned my_color = color[id];
	unsigned supervertex_id = new_vertex[my_color];
	unsigned edge = tex1Dfetch(tex_psrc, id);
	unsigned last_edge = edge + tex1Dfetch(tex_outdegree, id);

	for(; edge < last_edge; ++edge)
	{
		unsigned dst = tex1Dfetch(tex_edgessrcdst, edge);
		if(dst != g.nnodes)
		{
			unsigned other_color = color[dst];
			if(my_color != other_color)
			{
				unsigned top_edge = atomicInc(&(topedge_per_vertex[supervertex_id]), UINT_MAX);
				//next_edgessrcdst[top_edge] = other_supervertex;
				next_edgessrcdst[top_edge] = new_vertex[other_color];
				next_edgessrcwt[top_edge] = tex1Dfetch(tex_edgessrcwt, edge);
				new_map_edges[top_edge] = old_map_edges[edge];
			}
		}

	}
}

__global__
void load_weights(CSR_Graph g, unsigned int *selected_edges, unsigned int *vertex_minweight){
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id >= g.nedges + 1) return;

	if(selected_edges[id] == 1)
	{
		vertex_minweight[id] = g.edgessrcwt[id];
	}
}





MGPU_MEM(unsigned int) BoruvkaUMinho_GPU(CSR_Graph *h_graph, unsigned block_size){
	mgpu::ContextPtr context = mgpu::CreateCudaDevice(0, NULL, true);
	detect_devices();
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	//CSR_Graph *h_graph = new CSR_Graph(argv[1]);

	std::vector<CSR_Graph*> d_graph;
	d_graph.push_back(new CSR_Graph(h_graph->nnodes, h_graph->nedges, DEVICE));
	d_graph[0]->d_allocate();

	unsigned problem_size = h_graph->nnodes;
	unsigned edges_size = h_graph->nedges;
	//unsigned block_size = 1024;

	unsigned int *next_nnodes, *next_nedges;
	hipMalloc((void **)&next_nnodes, sizeof(unsigned int));
	hipMalloc((void **)&next_nedges, sizeof(unsigned int));

    unsigned int *d_changed, h_changed;
	if(hipMalloc((void **)&d_changed, sizeof(unsigned int)) != hipSuccess)
	{
		CudaTest(const_cast<char*>("allocating changed failed"));
	}

	MGPU_MEM(unsigned int) vertex_minedge = context->Malloc<unsigned int>(problem_size);
	MGPU_MEM(unsigned int) vertex_minweight = context->Fill<unsigned int>(edges_size+1, 0);
	MGPU_MEM(unsigned int) color = context->Malloc<unsigned int>(problem_size);	
	MGPU_MEM(unsigned int) new_vertex = context->Malloc<unsigned int>(problem_size);
	MGPU_MEM(unsigned int) supervertex_flag = context->Malloc<unsigned int>(problem_size);
	MGPU_MEM(unsigned int) topedge_per_vertex = context->Malloc<unsigned int>(problem_size);	
	MGPU_MEM(unsigned int) map_edges = context->FillAscending<unsigned int>(edges_size + 1, 0, 1);	
	MGPU_MEM(unsigned int) selected_edges = context->Fill<unsigned int>(edges_size + 1, 0);
	MGPU_MEM(unsigned int) new_map_edges = context->FillAscending<unsigned int>(edges_size + 1, 0, 1);	

	double starttime, endtime;
	float time;
	float timings[19];

	for(unsigned j = 0; j < 19; ++j) 
	{
		timings[j] = 0.0f;
	}

  	hipEvent_t start, stop;
	hipEventCreate(&start);  hipEventCreate(&stop);

	unsigned int iteration = 0;
	long unsigned int total_weight = 0;

	starttime = rtclock();

	hipEventRecord(start, 0);
	h_graph->copyHostToDevice(d_graph[0]); 
 	hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
 	timings[18] += time;
 	CudaTest(const_cast<char*>("copy CSR_Graph host to device failed"));

	do{
		//toString<<<1,1>>>(*d_graph[iteration]);
		unsigned n_blocks = compute_n_blocks(problem_size, block_size);
		printf("Graph has %u nodes and %u edges\n", problem_size, edges_size);

		//SegSortPairsFromIndices(d_graph[iteration]->edgessrcwt, d_graph[iteration]->edgessrcdst, edges_size+1, d_graph[iteration]->psrc, problem_size+1, *context);

		hipBindTexture(0, tex_psrc, d_graph[iteration]->psrc, sizeof(unsigned int) * problem_size);
		CudaTest(const_cast<char*>("bind tex_psrc failed"));
		hipBindTexture(0, tex_outdegree, d_graph[iteration]->outdegree, sizeof(unsigned int) * problem_size);
		CudaTest(const_cast<char*>("bind tex_outdegree failed"));
		hipBindTexture(0, tex_edgessrcdst, d_graph[iteration]->edgessrcdst, sizeof(unsigned int) * (edges_size + 1));
		CudaTest(const_cast<char*>("bind tex_edgessrcdst failed"));
		hipBindTexture(0, tex_edgessrcwt, d_graph[iteration]->edgessrcwt, sizeof(unsigned int) * (edges_size + 1));
		CudaTest(const_cast<char*>("bind tex_edgessrcwt failed"));


		hipEventRecord(start, 0);
		find_min_per_vertex<<<n_blocks, block_size>>>(*d_graph[iteration], vertex_minedge->get());
	 	hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
	 	timings[0] += time;
		CudaTest(const_cast<char*>("find_min_per_vertex failed"));


		// depends on find_min_per_vertex
		hipEventRecord(start, 0);
		remove_duplicates<<<n_blocks, block_size>>>(*d_graph[iteration], vertex_minedge->get());
		hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
		timings[16] += time;
		CudaTest(const_cast<char*>("remove_duplicates failed"));

		hipEventRecord(start, 0);
	 	initialize_color<<<n_blocks, block_size>>>(d_graph[iteration]->nnodes, color->get(), vertex_minedge->get());
	 	hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
	 	timings[1] += time;
	 	CudaTest(const_cast<char*>("initialize_color color failed"));

		do{
			hipEventRecord(start, 0);
			hipMemset(d_changed, 0, sizeof(unsigned int)); 
			hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
			timings[2] += time;
			CudaTest(const_cast<char*>("memset d_changed failed"));

			// depends on initialize color
			// depends on find_min_per_vertex
			hipEventRecord(start, 0);
			propagate_color<<<n_blocks, block_size>>>(d_graph[iteration]->nnodes, color->get(), d_changed);
			hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
			timings[3] += time;
			CudaTest(const_cast<char*>("propagate_color failed"));

			hipEventRecord(start, 0);	
			hipMemcpy(&h_changed, d_changed, sizeof(h_changed), hipMemcpyDeviceToHost);
			hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
			timings[4] += time;
			CudaTest(const_cast<char*>("copy d_changed failed"));
		} while(h_changed);

		///////////////////////
		// saving selected edges and weights
		///////////////////////


		hipEventRecord(start, 0);
		mark_mst_edges<<<n_blocks, block_size>>>(d_graph[iteration]->nnodes, selected_edges->get(), vertex_minedge->get(), map_edges->get());

		hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
		timings[17] += time;
		CudaTest(const_cast<char*>("mark_mst_edges failed"));

		///////////////////////
		// allocate new device graph
		///////////////////////
		hipEventRecord(start, 0);
		d_graph.push_back(new CSR_Graph(0, 0, DEVICE));
		hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
		timings[7] += time;
		CudaTest(const_cast<char*>("push_back failed"));

		///////////////////////
		// creating supervertices
		///////////////////////

		//depends on propagate colors
		hipEventRecord(start, 0);
		hipMemset(next_nnodes, 0, sizeof(unsigned int));
		create_new_vertex_id<<<n_blocks, block_size>>>(*d_graph[iteration], color->get(), supervertex_flag->get(), next_nnodes);
		CudaTest(const_cast<char*>("create_new_vertex_id failed"));
		mgpu::Scan<mgpu::MgpuScanTypeExc>(supervertex_flag->get(), problem_size, (unsigned int)0, mgpu::plus<unsigned int>(), (unsigned int*)0, &(d_graph[iteration+1]->nnodes), new_vertex->get(), *context);
		hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
		//hipMemcpy(&(d_graph[iteration+1]->nnodes), next_nnodes, sizeof(unsigned int), hipMemcpyDeviceToHost);
		timings[10] += time;
		CudaTest(const_cast<char*>("mgpu::scan failed"));
		

		unsigned new_nnodes;
		new_nnodes = d_graph[iteration+1]->nnodes;
		
		if(unlikely(new_nnodes <= 1))
		{
			hipUnbindTexture(tex_psrc);
			hipUnbindTexture(tex_outdegree);
			hipUnbindTexture(tex_edgessrcdst);
			hipUnbindTexture(tex_edgessrcwt);
				//if(iteration > 0) d_graph[iteration]->d_deallocate();
			d_graph[iteration]->d_deallocate();
			break;
		}

		d_graph[iteration+1]->d_allocate_nodes();
		hipDeviceSynchronize();
		///////////////////////
		// inserting new contracted edges
		///////////////////////

		// depends on propagate colors
		// depends on create_new_vertex_id
		hipEventRecord(start, 0);
		count_new_edges<<<n_blocks, block_size>>>(*d_graph[iteration], *d_graph[iteration+1], color->get(), new_vertex->get());
		hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
		timings[12] += time;
		CudaTest(const_cast<char*>("count_new_edges failed"));

		// depends on count_new_edges
		hipEventRecord(start, 0);
		hipMemset(next_nedges, 0, sizeof(unsigned int));
		mgpu::Scan<mgpu::MgpuScanTypeExc>(d_graph[iteration+1]->outdegree, new_nnodes, (unsigned int)0, mgpu::plus<unsigned int>(), (unsigned int*)0, &(d_graph[iteration+1]->nedges), d_graph[iteration+1]->psrc, *context);
		CudaTest(const_cast<char*>("mgpu::Scan failed"));
		setup_psrc<<<compute_n_blocks(new_nnodes, block_size), block_size>>>(*d_graph[iteration+1], next_nedges);
		CudaTest(const_cast<char*>("setup_psrc failed"));
		//hipMemcpy(&(d_graph[iteration+1]->nedges), next_nedges, sizeof(unsigned int), hipMemcpyDeviceToHost);
		hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
		timings[13] += time;

		d_graph[iteration+1]->d_allocate_edges();
		hipDeviceSynchronize();

		hipEventRecord(start, 0);
		hipMemcpy(topedge_per_vertex->get(), d_graph[iteration+1]->psrc, sizeof(unsigned int) * new_nnodes, hipMemcpyDeviceToDevice); 
		hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
		timings[14] += time;
		CudaTest(const_cast<char*>("copy topedge_per_vertex failed"));

		// depends on topedge_per_vertex memcpy
		// depends on setup_psrc
		hipEventRecord(start, 0);
		insert_new_edges<<<n_blocks, block_size>>>(*d_graph[iteration], d_graph[iteration+1]->edgessrcdst, d_graph[iteration+1]->edgessrcwt, color->get(), new_vertex->get(), topedge_per_vertex->get(), map_edges->get(), new_map_edges->get());
		hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
		timings[15] += time;
		CudaTest(const_cast<char*>("insert_new_edges failed"));
	
		edges_size = d_graph[iteration+1]->nedges;

		hipEventRecord(start, 0);		
		hipMemcpy(map_edges->get(), new_map_edges->get(), sizeof(unsigned int) * (edges_size + 1), hipMemcpyDeviceToDevice); 
		hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
		timings[6] += time;
		CudaTest(const_cast<char*>("copy map_edges failed"));

		problem_size = new_nnodes;

		hipUnbindTexture(tex_psrc);
		hipUnbindTexture(tex_outdegree);
		hipUnbindTexture(tex_edgessrcdst);
		hipUnbindTexture(tex_edgessrcwt);

		if(iteration > 0)
		{
			//hipDeviceSynchronize();
			d_graph[iteration]->d_deallocate();
			//hipDeviceSynchronize();
		}

		++iteration;

	} while(true);
	endtime = rtclock();

	// the selected MST edges are in the selected_edges array
	// hipEventRecord(start, 0);
	// load_weights<<<compute_n_blocks(h_graph->nedges + 1, block_size), block_size>>>(*d_graph[0], selected_edges->get(), vertex_minweight->get());	
	// mgpu::Reduce(vertex_minweight->get(), h_graph->nedges + 1, (long unsigned int)0, mgpu::plus<long unsigned int>(), (long unsigned int*)0, &total_weight, *context);
	// hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
	// timings[5] += time;
	// CudaTest(const_cast<char*>("mgpu::Reduce failed"));

	// unsigned int mst_edges = 0;
	// mgpu::Reduce(selected_edges->get(), h_graph->nedges + 1, (unsigned int)0, mgpu::plus<unsigned int>(), (unsigned int*)0, &mst_edges, *context);


	printf("%.1f\t ms on copying source graph to GPU\n", timings[18]);
	printf("%.1f\t ms on find_min_per_vertex\n", timings[0]);
	printf("%.1f\t ms on initialize_color\n", timings[1]);
	printf("%.1f\t ms on memset d_changed\n", timings[2]);
	printf("%.1f\t ms on propagate_color\n", timings[3]);
	printf("%.1f\t ms on copy d_changed\n", timings[4]);
	printf("%.1f\t ms on remove_duplicates\n", timings[16]);
	printf("%.1f\t ms on mark mst edges\n", timings[17]);
	printf("%.1f\t ms on push_back\n", timings[7]);
	printf("%.1f\t ms on create_new_vertex_id\n", timings[10]);
	printf("%.1f\t ms on count_new_edges\n", timings[12]);
	printf("%.1f\t ms on setup_psrc\n", timings[13]);
	printf("%.1f\t ms on copy topedge_per_vertex\n", timings[14]);
	printf("%.1f\t ms on insert_new_edges\n", timings[15]);
	printf("%.1f\t ms on copy map_edges\n", timings[6]);
	printf("%.3lf\t ms total execution time\n", 1000 * (endtime - starttime));

	//printf("\t%.1f ms on weight computation\n", timings[5]);

	//printf("total mst weight %lu (not counting mirrored edges (/2): %lu) and %u edges\n", total_weight*2, total_weight, mst_edges-1);

	return selected_edges;
}
