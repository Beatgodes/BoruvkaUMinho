#include "hip/hip_runtime.h"
#include "BoruvkaUMinho_GPU.cuh"

__global__
void load_weights(CSR_Graph g, unsigned int *selected_edges, unsigned int *vertex_minweight){
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id >= g.nedges + 1) return;

	if(selected_edges[id] == 1)
	{
		vertex_minweight[id] = g.edgessrcwt[id];
	}
}

int main(int argc, char *argv[]){
	if(argc != 2)
	{
		printf("Wrong nr of args\n");
		return 1;
	}

	mgpu::ContextPtr context = mgpu::CreateCudaDevice(0, NULL, true);

	CSR_Graph *g = new CSR_Graph(argv[1]);
	unsigned block_size = atoi(argv[2]);

	MGPU_MEM(unsigned int) selected_edges = BoruvkaUMinho_GPU(g, block_size);

	long unsigned int total_weight = 0;
	MGPU_MEM(unsigned int) vertex_minweight = context->Fill<unsigned int>(g->nedges+1, 0);

	load_weights<<<compute_n_blocks(g->nedges + 1, block_size), block_size>>>(*d_graph[0], selected_edges->get(), vertex_minweight->get());	
	mgpu::Reduce(vertex_minweight->get(), g->nedges + 1, (long unsigned int)0, mgpu::plus<long unsigned int>(), (long unsigned int*)0, &total_weight, *context);
	CudaTest(const_cast<char*>("mgpu::Reduce failed"));

	unsigned int mst_edges = 0;
	mgpu::Reduce(selected_edges->get(), g->nedges + 1, (unsigned int)0, mgpu::plus<unsigned int>(), (unsigned int*)0, &mst_edges, *context);
	CudaTest(const_cast<char*>("mgpu::Reduce 2 failed"));

	printf("total mst weight %lu (not counting mirrored edges (/2): %lu) and %u edges\n", total_weight*2, total_weight, mst_edges-1);
	

	selected_edges->Free();
	vertex_minweight->Free();

	return 0;
}
